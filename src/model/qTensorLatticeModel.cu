#include "hip/hip_runtime.h"
#include "qTensorLatticeModel.cuh"
#include "../../inc/qTensorFunctions.h"
//#incldue "qTensorfunctions.h"
/*! \file qTensorLatticeModel.cu */

/*!
    \addtogroup modelKernels
    @{
*/

__global__ void gpu_largestEigenvalue_kernel(dVec *Q,scalar *defects,int *t, int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    if(t[idx]>0)
        return;
    scalar a,b,c;
    eigenvaluesOfQ(Q[idx],a,b,c);
    defects[idx] = max(max(a,b),c);
    return;
    }


__global__ void gpu_computeDeterminant_kernel(dVec *Q,scalar *defects,int *t, int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    if(t[idx]>0)
        return;
    defects[idx] = determinantOfQ(Q[idx]);
    return;
    }

__global__ void gpu_degenerateEigenvalue_kernel(dVec *Q,scalar *defects,int *t, int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    if(t[idx]>0)
        return;
    scalar trQ2 = TrQ2(Q[idx]);
    scalar det = determinantOfQ(Q[idx]);
    defects[idx] = trQ2*trQ2*trQ2 - 54.0*det*det;
    return;
    }

__global__ void gpu_set_random_nematic_qTensors_kernel(dVec *pos, int *type, hiprandState *rngs,scalar amplitude, bool globallyAligned, scalar globalTheta, scalar globalPhi,int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    hiprandState randState;
    randState = rngs[idx];

    scalar theta = acos(2.0*hiprand_uniform(&randState)-1);
    scalar phi = 2.0*PI*hiprand_uniform(&randState);
    if(globallyAligned)
        {
        theta = globalTheta;
        phi = globalPhi;
        }
    scalar3 n;
    n.x = cos(phi)*sin(theta);
    n.y = sin(phi)*sin(theta);
    n.z = cos(theta);

    if(type[idx] <=0)
        qTensorFromDirector(n, amplitude, pos[idx]);
    rngs[idx] = randState;
    return;
    };

__global__ void gpu_update_qTensor_simple_kernel(dVec *d_disp,
                            dVec *d_pos,
                            scalar scale,
                            int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int pidx = idx/DIMENSION;
    if(pidx>=N) return;
    int didx = idx%DIMENSION;

    d_pos[pidx][didx] += scale*d_disp[pidx][didx];
    /*
    scalar max = (didx >2 ) ? .5 : .834;
    scalar min = (didx >2 ) ? -.75 : -.667;
    if(d_pos[pidx][didx] > max)
        d_pos[pidx][didx] = max;
    if(d_pos[pidx][didx] < min)
        d_pos[pidx][didx] = min;
    */    
    return;
    };

__global__ void gpu_update_qTensor_simple_kernel(dVec *d_disp,
                            dVec *d_pos,
                            int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int pidx = idx/DIMENSION;
    if(pidx>=N) return;
    int didx = idx%DIMENSION;

    d_pos[pidx][didx] += d_disp[pidx][didx];
    /*
    scalar max = (didx >2 ) ? .5 : .834;
    scalar min = (didx >2 ) ? -.75 : -.667;
    if(d_pos[pidx][didx] > max)
        d_pos[pidx][didx] = max;
    if(d_pos[pidx][didx] < min)
        d_pos[pidx][didx] = min;
    */    
    return;
    };

bool gpu_update_qTensor(dVec *d_disp,
                            dVec *Q,
                            scalar scale,
                            int N,int blockSize)
    {
    if (N < 128) blockSize = 16;
    unsigned int nBlocks  = DIMENSION*N/blockSize + 1;
    if(DIMENSION <5)
        {
        printf("\nAttempting to initialize Q-tensors with incorrectly set dimension...change the root CMakeLists.txt file to have dimension 5 and recompile\n");
        throw std::exception();
        }
    gpu_update_qTensor_simple_kernel<<<nBlocks,blockSize>>>(d_disp, Q,scale,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_update_qTensor(dVec *d_disp,
                            dVec *Q,
                            int N,int blockSize)
    {
    if (N < 128) blockSize = 16;
    unsigned int nBlocks  = DIMENSION*N/blockSize + 1;
    if(DIMENSION <5)
        {
        printf("\nAttempting to initialize Q-tensors with incorrectly set dimension...change the root CMakeLists.txt file to have dimension 5 and recompile\n");
        throw std::exception();
        }
    gpu_update_qTensor_simple_kernel<<<nBlocks,blockSize>>>(d_disp, Q,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

bool gpu_set_random_nematic_qTensors(dVec *d_pos,
                          int *d_types,
                          hiprandState *rngs,
                          scalar amplitude,
                          int blockSize,
                          int nBlocks,
                          bool globallyAligned,
                          scalar theta,
                          scalar phi,
                          int N
                          )
    {
    if(DIMENSION <5)
        {
        printf("\nAttempting to initialize Q-tensors with incorrectly set dimension...change the root CMakeLists.txt file to have dimension 5 and recompile\n");
        throw std::exception();
        }
    gpu_set_random_nematic_qTensors_kernel<<<nBlocks,blockSize>>>(d_pos,d_types, rngs,amplitude, globallyAligned, theta, phi,N);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }


bool gpu_get_qtensor_DefectMeasures(dVec *Q,
                                    scalar *defects,
                                    int *t,
                                    int defectType,
                                    int N)
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (N < 128) block_size = 16;
    unsigned int nblocks  = N/block_size + 1;
    if(defectType ==0)
        gpu_largestEigenvalue_kernel<<<nblocks,block_size>>>(Q,defects,t,N);
    if(defectType ==1)
        gpu_computeDeterminant_kernel<<<nblocks,block_size>>>(Q,defects,t,N);
    if(defectType ==2)
        gpu_degenerateEigenvalue_kernel<<<nblocks,block_size>>>(Q,defects,t,N);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
